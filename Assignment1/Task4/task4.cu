#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <limits>

#include <sys/time.h>
#include <time.h>

#include <hip/hip_runtime.h>

using namespace std;

__global__ void assignmentKernel(float* d_in, float* d_out, int threads_num) {
  const unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if(tid < threads_num) {
    float x = d_in[tid];
    d_out[tid] = pow((x/(x-2.3)),3.0);
  }
}

int timeval_subtract( struct timeval* result, struct timeval* t2,struct timeval* t1) {
  unsigned int resolution=1000000;
  long int diff = (t2->tv_usec + resolution * t2->tv_sec) -
    (t1->tv_usec + resolution * t1->tv_sec) ;
  result->tv_sec = diff / resolution;
  result->tv_usec = diff % resolution;
  return (diff<0);
}

unsigned long int cudaCalculation(unsigned int num_threads, unsigned int mem_size, float* h_in, float* h_out) {
  unsigned int block_size  = 1024;
  unsigned int num_blocks  = ceil(((num_threads + (block_size -1)) / block_size));

  // Allocate device memory
  float* d_in;
  hipMalloc((void**) &d_in,  mem_size);
  float* d_out;
  hipMalloc((void**) &d_out, mem_size);

  // Copy host memory to device
  hipMemcpy(d_in, h_in, mem_size, hipMemcpyHostToDevice);

  // Start timing the CUDA run.
  unsigned long int elapsed;
  struct timeval t_start, t_end, t_diff;
  gettimeofday(&t_start, NULL);

  // Execute the kernel
  assignmentKernel<<<num_blocks, block_size>>>(d_in, d_out, num_threads);
  hipDeviceSynchronize();

  // End timing the CUDA run,
  gettimeofday(&t_end, NULL);
  timeval_subtract(&t_diff, &t_end, &t_start);
  elapsed = t_diff.tv_sec*1e6+t_diff.tv_usec;

  // Copy the result form device to host
  hipMemcpy(h_out, d_out, mem_size, hipMemcpyDeviceToHost);

  // Free device memory.
  hipFree(d_in);
  hipFree(d_out);

  return elapsed;
}

unsigned long int cpuCalculation(unsigned int array_length, float* h_in, float* h_out) {

  // Start timing the CPU run.
  unsigned long int elapsed;
  struct timeval t_start, t_end, t_diff;
  gettimeofday(&t_start, NULL);

  // Perform serial calculations
  for (unsigned int i=0 ; i < array_length ; i++) {
      float x = h_in[i];
      h_out[i] = pow((x/(x-2.3)),3.0);
  }

  // End timing the CPU run,
  gettimeofday(&t_end, NULL);
  timeval_subtract(&t_diff, &t_end, &t_start);
  elapsed = t_diff.tv_sec*1e6+t_diff.tv_usec;

  return elapsed;
}

bool areSame(float* h_in, float* cpu_arr, float* cuda_arr, unsigned int array_length) {
  for(unsigned int i = 0 ; i < array_length ; i++) {
    if(!(std::abs(cpu_arr[i] - cuda_arr[i]) < std::numeric_limits<float>::epsilon())) {
      printf("INVALID:\nInput: %f\nCPU: %.15f\nGPU: %.15f\n", h_in[i], cpu_arr[i], cuda_arr[i]);
      return false;
    }
  }
  return true;
}

int main(int argc, char** argv) {
  unsigned int array_length = 753411;
  unsigned int mem_size   = array_length*sizeof(float);

  // Allocate host memory
  float* h_in  = (float*) malloc(mem_size);
  float* cuda_out = (float*) malloc(mem_size);
  float* cpu_out = (float*) malloc(mem_size);

  // Initialize the memory
  for(unsigned int i=0 ; i < array_length ; ++i) {
    h_in[i] = (float)i+1;
  }

  // Call the CUDA code.
  unsigned long int cuda_elapsed = cudaCalculation(array_length, mem_size, h_in, cuda_out);
  printf("CUDA took %d microseconds (%.2fms)\n",cuda_elapsed,cuda_elapsed/1000.0);

  // Call the CPU code.
  unsigned long int cpu_elapsed = cpuCalculation(array_length, h_in, cpu_out);
  printf("CPU took %d microseconds (%.2fms)\n",cpu_elapsed,cpu_elapsed/1000.0);

  if(areSame(h_in, cpu_out, cuda_out, array_length)) {
    printf("VALID: CPU and GPU agreed on results.\n");
  }

  // print result
  //for(unsigned int i=0; i<array_length; ++i) printf("%.6f\n", cuda_out[i]);

  // Clean up memory
  free(h_in);
  free(cuda_out);
  free(cpu_out);

  return 0;
}
